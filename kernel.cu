﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<hipsparse.h>
#include<vector>
#include<iostream>
using namespace std;

void checkCudaError(hipError_t err, const char* msg)
{
	if (err != hipSuccess)
	{
		cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << endl;
		exit(EXIT_FAILURE);
	}
}

void checkCusparseError(hipsparseStatus_t status, const char* msg)
{
	if (status != HIPSPARSE_STATUS_SUCCESS)
	{
		cerr << "cuSPARSE error: " << msg << endl;
		exit(EXIT_FAILURE);
	}
}
int main() {
    // Example matrix in dense form (4x4)
    std::vector<float> denseMatrix = {
        1.0, 0.0, 2.0, 0.0,
        0.0, 3.0, 0.0, 0.0,
        0.0, 0.0, 4.0, 5.0,
        0.0, 6.0, 0.0, 0.0
    };
    const int rows = 4, cols = 4;
    const int nnz = 6; // Number of non-zero elements

    // Dense vector
    std::vector<float> denseVector = { 1.0, 2.0, 3.0, 4.0 };

    // Result vector
    std::vector<float> result(rows, 0.0);

    // Device memory pointers
    float* d_denseMatrix, * d_denseVector, * d_result;
    hipsparseHandle_t cusparseHandle;
    hipsparseMatDescr_t matDescr;
    hipsparseHybMat_t hybMatrix;

    // Allocate memory on device
    checkCudaError(hipMalloc((void**)&d_denseMatrix, rows * cols * sizeof(float)), "hipMalloc denseMatrix");
    checkCudaError(hipMalloc((void**)&d_denseVector, cols * sizeof(float)), "hipMalloc denseVector");
    checkCudaError(hipMalloc((void**)&d_result, rows * sizeof(float)), "hipMalloc result");

    // Copy data to device
    checkCudaError(hipMemcpy(d_denseMatrix, denseMatrix.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy denseMatrix");
    checkCudaError(hipMemcpy(d_denseVector, denseVector.data(), cols * sizeof(float), hipMemcpyHostToDevice), "hipMemcpy denseVector");

    // Initialize cuSPARSE
    checkCusparseError(hipsparseCreate(&cusparseHandle), "hipsparseCreate");
    checkCusparseError(hipsparseCreateMatDescr(&matDescr), "hipsparseCreateMatDescr");
    checkCusparseError(hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL), "hipsparseSetMatType");
    checkCusparseError(hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO), "hipsparseSetMatIndexBase");

    // Create HYB matrix
    checkCusparseError(hipsparseCreateHybMat(&hybMatrix), "hipsparseCreateHybMat");

    // Convert dense matrix to HYB format
    checkCusparseError(
        cusparseSdense2hyb(cusparseHandle, rows, cols, matDescr, d_denseMatrix, rows, hybMatrix, 0, HIPSPARSE_HYB_PARTITION_AUTO),
        "cusparseSdense2hyb"
    );

    // Perform SpMV (HYB format)
    const float alpha = 1.0f, beta = 0.0f;
    checkCusparseError(
        hipsparseShybmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matDescr, hybMatrix, d_denseVector, &beta, d_result),
        "hipsparseShybmv"
    );

    // Copy result back to host
    checkCudaError(hipMemcpy(result.data(), d_result, rows * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy result");

    // Print result
    std::cout << "Result vector: ";
    for (const auto& val : result) {
        std::cout << val << " ";
    }
    std::cout << std::endl;

    // Clean up
    checkCudaError(hipFree(d_denseMatrix), "hipFree denseMatrix");
    checkCudaError(hipFree(d_denseVector), "hipFree denseVector");
    checkCudaError(hipFree(d_result), "hipFree result");
    checkCusparseError(hipsparseDestroyHybMat(hybMatrix), "hipsparseDestroyHybMat");
    checkCusparseError(hipsparseDestroyMatDescr(matDescr), "hipsparseDestroyMatDescr");
    checkCusparseError(hipsparseDestroy(cusparseHandle), "hipsparseDestroy");

    return 0;
}